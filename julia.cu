#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>

#define DIM 1000

struct hipComplex {
    float r;
    float i;

    __host__ __device__ hipComplex(float a, float b) : r(a), i(b) {}

    __host__ __device__ float magnitude2(void) {
        return r * r + i * i;
    }

    __host__ __device__ hipComplex operator*(const hipComplex& a) {
        return hipComplex(r * a.r - i * a.i, i * a.r + r * a.i);
    }

    __host__ __device__ hipComplex operator+(const hipComplex& a) {
        return hipComplex(r + a.r, i + a.i);
    }
};

__device__ int julia(int x, int y) {
    const float scale = 1.5;
    float jx = scale * (float)(DIM / 2 - x) / (DIM / 2);
    float jy = scale * (float)(DIM / 2 - y) / (DIM / 2);
    hipComplex c(-0.8, 0.156);
    hipComplex a(jx, jy);
    int i = 0;

    for (i = 0; i < 200; i++) {
        a = a * a + c;
        if (a.magnitude2() > 1000)
            return 0;
    }
    return 1;
}

__global__ void kernel(unsigned char *ptr) {
    int x = blockIdx.x;
    int y = blockIdx.y;
    int offset = x + y * gridDim.x;

    int juliaValue = julia(x, y);
    ptr[offset * 4 + 0] = 255 * juliaValue;
    ptr[offset * 4 + 1] = 0;
    ptr[offset * 4 + 2] = 0;
    ptr[offset * 4 + 3] = 255;
}

void saveBMP(const unsigned char *bitmap, int width, int height, const char *filename) {
    std::ofstream ofs(filename, std::ios::out | std::ios::binary);

    if (!ofs) {
        std::cerr << "Could not open file for writing: " << filename << std::endl;
        return;
    }

    unsigned int fileSize = 54 + width * height * 4;
    unsigned char fileHeader[14] = {
        'B', 'M', // BMP signature
        0, 0, 0, 0, // File size in bytes
        0, 0, 0, 0, // Reserved
        54, 0, 0, 0 // Start of pixel data
    };

    unsigned char infoHeader[40] = {
        40, 0, 0, 0, // Info header size
        0, 0, 0, 0, // Width
        0, 0, 0, 0, // Height
        1, 0,       // Number of color planes
        32, 0,      // Bits per pixel
        0, 0, 0, 0, // Compression
        0, 0, 0, 0, // Image size (no compression)
        0, 0, 0, 0, // Horizontal resolution
        0, 0, 0, 0, // Vertical resolution
        0, 0, 0, 0, // Number of colors
        0, 0, 0, 0  // Important colors
    };

    fileHeader[2] = (unsigned char)(fileSize);
    fileHeader[3] = (unsigned char)(fileSize >> 8);
    fileHeader[4] = (unsigned char)(fileSize >> 16);
    fileHeader[5] = (unsigned char)(fileSize >> 24);

    infoHeader[4] = (unsigned char)(width);
    infoHeader[5] = (unsigned char)(width >> 8);
    infoHeader[6] = (unsigned char)(width >> 16);
    infoHeader[7] = (unsigned char)(width >> 24);

    infoHeader[8] = (unsigned char)(height);
    infoHeader[9] = (unsigned char)(height >> 8);
    infoHeader[10] = (unsigned char)(height >> 16);
    infoHeader[11] = (unsigned char)(height >> 24);

    ofs.write(reinterpret_cast<const char *>(fileHeader), sizeof(fileHeader));
    ofs.write(reinterpret_cast<const char *>(infoHeader), sizeof(infoHeader));
    ofs.write(reinterpret_cast<const char *>(bitmap), width * height * 4);

    ofs.close();
}

int main(void) {
    unsigned char *dev_bitmap;
    unsigned char *bitmap = new unsigned char[DIM * DIM * 4];

    hipMalloc((void**)&dev_bitmap, DIM * DIM * 4);
    dim3 grid(DIM, DIM);
    kernel<<<grid, 1>>>(dev_bitmap);
    hipMemcpy(bitmap, dev_bitmap, DIM * DIM * 4, hipMemcpyDeviceToHost);
    hipFree(dev_bitmap);

    saveBMP(bitmap, DIM, DIM, "julia.bmp");

    delete[] bitmap;

    return 0;
}



